#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "mytime.h"

//GPU kernel
__global__ void integrate(int *n, double *sum){
	double h, x;
	int i;
	
	*sum = 0.0;
	h = 1.0/(double) *n;
	for(i=1; i<=*n, i++){
		x = h * ((double)i - 0.5);
		*sum += 4.0 / (1.0 + x*x);
	}
	*sum *= h;
}

int main(int argc, char *argv[]){
	int n;
	int *n_d;
	double pi16 = 3.1415926535897932;
	double pi, pi_d;
	
	//Allocate memory on GPU
	hipMalloc((void**) &n_d, sizeof(int) * 1);
	hipMalloc((void**) &pi_d, sizeof(double) * 1);
	
	while(1){
		printf("\nEnter the number of intervals: ");
		scanf("%n");
		if(n == 0){
			break;
		}
		//copy from CPU to GPU
		hipMemcpy(n_d, &n, sizeof(int) * 1, hipMemcpyHostToDevice);
		integrate<<<1,1>>>(n_d, pi_d);
		//copy back from GPU to CPU
		cudeMemcpy(&pi, pi_d, sizeof(double) * 1, cudaMemcptDeviceToHost);
		printf("\npi=%.16f, error=%.16f", pi, fabs(pi-pi16));
	}
	
	hipFree(n_d)
	hipFree(pi_d);
}